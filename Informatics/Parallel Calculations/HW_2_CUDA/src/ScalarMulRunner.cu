#include <CommonKernels.cuh>
#include <KernelMul.cuh>
#include <ScalarMul.cuh>
#include <ScalarMulRunner.cuh>

float ScalarMulTwoReductions(unsigned size, const float* lhs, const float* rhs, unsigned block_size) {
  unsigned block1_cnt = (size + block_size - 1) / block_size;
  unsigned block2_cnt = (block1_cnt + block_size - 1) / block_size;

  float* lhs_cuda;
  float* rhs_cuda;
  float* result1_cuda;
  float* result2_cuda;
  auto* result = new float[block2_cnt];

  hipMalloc(&lhs_cuda, size * sizeof(float));
  hipMalloc(&rhs_cuda, size * sizeof(float));
  hipMalloc(&result1_cuda, size * sizeof(float));
  hipMalloc(&result2_cuda, block1_cnt * sizeof(float));

  hipMemcpy(lhs_cuda, lhs, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(rhs_cuda, rhs, size * sizeof(float), hipMemcpyHostToDevice);

  float ans = 0;
  KernelMul<<<block1_cnt, block_size>>>(size, lhs_cuda, rhs_cuda, result2_cuda);
  hipDeviceSynchronize();
  ReduceWrapperIDs<<<block1_cnt, block_size, block_size * sizeof(float)>>>(size, result1_cuda, result2_cuda);
  hipDeviceSynchronize();
  ReduceWrapperIDs<<<block2_cnt, block_size, block_size * sizeof(float)>>>(block1_cnt, result2_cuda, result1_cuda);
  hipDeviceSynchronize();

  hipMemcpy(result, result2_cuda, block2_cnt * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < block2_cnt; ++i)
    ans += result[i];

  delete[] result;

  hipFree(lhs_cuda);
  hipFree(rhs_cuda);
  hipFree(result1_cuda);
  hipFree(result2_cuda);

  return ans;
}

float ScalarMulSumPlusReduction(unsigned size, const float* lhs, const float* rhs, unsigned block_size) {
  unsigned block1_cnt = (size + block_size - 1) / block_size;

  float* lhs_cuda;
  float* rhs_cuda;
  float* result_cuda;

  hipMalloc(&lhs_cuda, size * sizeof(float));
  hipMalloc(&rhs_cuda, size * sizeof(float));
  hipMalloc(&result_cuda, block1_cnt * sizeof(float));

  hipMemcpy(lhs_cuda, lhs, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(rhs_cuda, rhs, size * sizeof(float), hipMemcpyHostToDevice);

  float ans = 0;
  ScalarMulBlock<<<block1_cnt, block_size, block_size * sizeof(float)>>>(size, lhs_cuda, rhs_cuda, result_cuda);
  hipDeviceSynchronize();
  ReduceWrapperIDs<<<1, block_size, block_size * sizeof(float)>>>(block1_cnt, result_cuda, result_cuda);
  hipDeviceSynchronize();
  hipMemcpy(&ans, result_cuda, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(lhs_cuda);
  hipFree(rhs_cuda);
  hipFree(result_cuda);

  return ans;
}
